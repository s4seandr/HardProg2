#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>

#define BLOCK_SIZE 1024
#define FFT_SIZE 1024
#define HOP_SIZE 64
#define AMPLITUDE_THRESHOLD 0.1

typedef struct {
    char chunkId[4];
    int chunkSize;
    char format[4];
    char subchunk1Id[4];
    int subchunk1Size;
    short audioFormat;
    short numChannels;
    int sampleRate;
    int byteRate;
    short blockAlign;
    short bitsPerSample;
    char subchunk2Id[4];
    int subchunk2Size;
} WavHeader;

__global__ void convertToDouble(short* input, double* output, int start, int numSamples) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numSamples) {
        output[idx] = input[start + idx] / ((double)SHRT_MAX);
    }
}

__global__ void calculateAmplitude(hipfftDoubleComplex* input, double* output, int numSamples) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numSamples / 2) { // Nur die erste Hälfte der FFT-Ergebnisse ist relevant
        output[idx] = sqrt(input[idx].x * input[idx].x + input[idx].y * input[idx].y) / (numSamples / 2);
    }
}

int main(int argc, char** argv) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Startzeit aufzeichnen
    hipEventRecord(start, 0);

    char* wavFilePath = "C:\\Users\\Sebi\\IdeaProjects\\untitled1\\output.wav";

    FILE* file = fopen(wavFilePath, "rb");
    if (file == NULL) {
        printf("Could not open WAV file\n");
        return 1;
    }

    WavHeader header;
    fread(&header, sizeof(WavHeader), 1, file);

    int numSamples = header.subchunk2Size / 2;
    short* samples = (short*)malloc(numSamples * sizeof(short));
    for (int i = 0; i < numSamples; i++) {
        fread(&samples[i], sizeof(short), 1, file);
    }
    fclose(file);

    short* d_samples;
    hipMalloc((void**)&d_samples, numSamples * sizeof(short));
    hipMemcpy(d_samples, samples, numSamples * sizeof(short), hipMemcpyHostToDevice);

    double* d_samplesDouble;
    hipMalloc((void**)&d_samplesDouble, FFT_SIZE * sizeof(double));

    hipfftHandle plan;
    hipfftDoubleComplex* d_fft;
    hipMalloc((void**)&d_fft, FFT_SIZE * sizeof(hipfftDoubleComplex));
    hipfftPlan1d(&plan, FFT_SIZE, HIPFFT_D2Z, 1);

    double* d_amplitude;
    hipMalloc((void**)&d_amplitude, FFT_SIZE / 2 * sizeof(double)); // Nur die erste Hälfte ist relevant

    double* sum = (double*)calloc(FFT_SIZE / 2, sizeof(double));
    double* count = (double*)calloc(FFT_SIZE / 2, sizeof(double));

    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    dim3 dimGrid((FFT_SIZE - 1) / BLOCK_SIZE + 1, 1, 1);

    double* amplitude = (double*)malloc(FFT_SIZE / 2 * sizeof(double));

    for (int i = 0; i < numSamples - FFT_SIZE; i += HOP_SIZE) {
        convertToDouble << <dimGrid, dimBlock >> > (d_samples, d_samplesDouble, i, FFT_SIZE);
        hipfftExecD2Z(plan, d_samplesDouble, d_fft);

        calculateAmplitude << <dimGrid, dimBlock >> > (d_fft, d_amplitude, FFT_SIZE);
        hipMemcpy(amplitude, d_amplitude, FFT_SIZE / 2 * sizeof(double), hipMemcpyDeviceToHost);

        for (int j = 0; j < FFT_SIZE / 2; j++) {
            sum[j] += amplitude[j];
            count[j]++;
        }
    }

    // Stoppzeit aufzeichnen
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int j = 0; j < FFT_SIZE / 2; j++) {
        double average = sum[j] / count[j];
        if (average > AMPLITUDE_THRESHOLD) {
            double frequency = (double)j * header.sampleRate / FFT_SIZE;
            printf("Frequenz: %f Hz, Durchschnittliche Amplitude: %f\n", frequency, average);
        }
    }
    printf("Programm dauerte %f Sekunden.\n", milliseconds / 1000);

    free(samples);
    free(sum);
    free(count);
    free(amplitude);
    hipFree(d_samples);
    hipFree(d_samplesDouble);
    hipFree(d_fft);
    hipFree(d_amplitude);
    hipfftDestroy(plan);

    return 0;
}
